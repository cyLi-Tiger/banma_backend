#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>
#include <names.h>
#include <km.h>
#include <vector>

namespace triton { namespace backend { namespace NAMESPACE {

namespace {
__global__ void km_kernel(float* vec1, float* vec2, float* cost, 
                          float const* matrix, float epsilon) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < size) {
        vec1[id] = matrix[id];
        vec2[id] = matrix[id];
        atomicAdd(cost, matrix[id]);
    }    
}
}

// void km(float* vec1, float* vec2, float cost, 
//         float const* matrix, float epsilon) {

// }



const int size = 32;

int main() {
    std::vector<std::vector<float>> inp(size, std::vector<float>(size, 6));
    float *h_vec1 = new float[size];
    float *h_vec2 = new float[size];
    float *h_cost = new float[1];

    float* matrix, *vec1, *vec2, *cost;
    hipMalloc((float**)&matrix, size * size * sizeof(float));
    hipMalloc((float**)&vec1, size * sizeof(float));
    hipMalloc((float**)&vec2, size * sizeof(float));
    hipMalloc((float**)&cost, 1 * sizeof(float));
    
    for (int i = 0; i < size; i++) {
        hipMemcpy(matrix + i * size, &inp[i][0], size * sizeof(float), hipMemcpyHostToDevice);
    }
    int block_size = 32;
    int grid_size = size * size / block_size;
    km_kernel<<<block_size, grid_size>>>(vec1, vec2, cost, matrix, 1e-4);
    hipMemcpy(h_vec1, vec1, size * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(h_vec2, vec2, size * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(h_cost, cost, 1 * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(matrix);
    hipFree(vec1);
    hipFree(vec2);
    hipFree(cost);
    delete[] h_vec1;
    delete[] h_vec2;
    delete[] h_cost;
}

}}}